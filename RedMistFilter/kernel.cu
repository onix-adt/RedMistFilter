#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include "lodepng.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void getError(hipError_t err) {
	if (err != hipSuccess) {
		std::cout << "CUDA Error " << hipGetErrorString(err) << std::endl;
	}
}

__global__ void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

	const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
	int x = offset % width;
	int y = (offset - x) / width;
	int fsize = 5; // Filter size
	if (offset < width*height) {

		float output_red = 0;
		float output_green = 0;
		float output_blue = 0;
		int hits = 0;
		for (int ox = -fsize; ox < fsize + 1; ++ox) {
			for (int oy = -fsize; oy < fsize + 1; ++oy) {
				if ((x + ox) > -1 && (x + ox) < width && (y + oy) > -1 && (y + oy) < height) {
					const int currentoffset = (offset + ox + oy*width) * 3;
					output_red += input_image[currentoffset];
					output_green += input_image[currentoffset + 1];
					output_blue += input_image[currentoffset + 2];
					hits++;
				}
			}
		}
		output_image[offset * 3] = output_red / hits;
		output_image[offset * 3 + 1] = output_green / hits;
		output_image[offset * 3 + 2] = output_blue / hits;
	}
}

void filter(unsigned char* input_image, unsigned char* output_image, int width, int height) {

	unsigned char* dev_input;
	unsigned char* dev_output;
	getError(hipMalloc((void**)&dev_input, width*height * 3 * sizeof(unsigned char)));
	getError(hipMemcpy(dev_input, input_image, width*height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

	getError(hipMalloc((void**)&dev_output, width*height * 3 * sizeof(unsigned char)));

	dim3 blockDims(512, 1, 1);
	dim3 gridDims((unsigned int)ceil((double)(width*height * 3 / blockDims.x)), 1, 1);

	blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);

	getError(hipMemcpy(output_image, dev_output, width*height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

	getError(hipFree(dev_input));
	getError(hipFree(dev_output));

}

int main(int argc, char** argv)
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	if (argc != 3) {
		std::cout << "Enter two command line params: input.png output.png" << std::endl;
		return 0;
	}

	const char* input_file = argv[1];
	const char* output_file = argv[2];

	std::vector<unsigned char> in_image;
	unsigned int width, height;

	// Load the data
	unsigned error = lodepng::decode(in_image, width, height, input_file);
	if (error) {
		std::cout << "PNG decoder error " << error << ": " << lodepng_error_text(error) << std::endl;
		return 1;
	}

	int size_rgb = (in_image.size() / 4) * 3;

	// Prepare data for CUDA 
	unsigned char* input_image = new unsigned char[size_rgb];
	unsigned char* output_image = new unsigned char[size_rgb];
	int index = 0;
	for (int i = 0; i < in_image.size(); ++i) {
		if (i % 4 != 0) {
			input_image[index] = in_image.at(i);
			index++;
		}
	}

	// Filtering
	filter(input_image, output_image, width, height);

	// Prepare data for output
	std::vector<unsigned char> out_image;
	for (int i = 0; i < size_rgb; ++i) {
		if (i % 3 == 0) {
			out_image.push_back(255);
		}
		out_image.push_back(output_image[i]);
	}
	out_image.push_back(255);

	// Output the data
	error = lodepng::encode(output_file, out_image, width, height);

	//if there's an error, display it
	if (error) {
		std::cout << "PNG encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
	}

	// Clean up
	delete[] input_image;
	delete[] output_image;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
